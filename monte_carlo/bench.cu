#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Constants
constexpr int NUM_PATHS = 1'000'000;
constexpr int NUM_STEPS = 1000;
constexpr float S0 = 100.0f;
constexpr float mu = 0.05f;
constexpr float sigma = 0.2f;
constexpr float T = 1.0f;
constexpr int BLOCK_SIZE = 256;

// CUDA error checking macro
#define CUDA_CHECK(call) \
do { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                  << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

__global__ void gbm_kernel(float* d_paths, float dt, hiprandState* states) {
    const int path_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (path_idx >= NUM_PATHS) return;

    hiprandState local_state = states[path_idx];
    float S = S0;
    d_paths[path_idx * (NUM_STEPS + 1)] = S0;

    for (int step = 0; step < NUM_STEPS; step++) {
        float z = hiprand_normal(&local_state);
        S *= expf((mu - 0.5f * sigma * sigma) * dt + sigma * sqrtf(dt) * z);
        d_paths[path_idx * (NUM_STEPS + 1) + step + 1] = S;
    }
    states[path_idx] = local_state;
}

void run_benchmark() {
    // Allocate device memory
    float* d_paths;
    CUDA_CHECK(hipMalloc(&d_paths, NUM_PATHS * (NUM_STEPS + 1) * sizeof(float)));

    // Allocate and initialize RNG states
    hiprandState* d_states;
    CUDA_CHECK(hipMalloc(&d_states, NUM_PATHS * sizeof(hiprandState)));

    // Setup RNG
    const int blocks = (NUM_PATHS + BLOCK_SIZE - 1) / BLOCK_SIZE;
    curandSetupKernel<<<blocks, BLOCK_SIZE>>>(d_states, time(nullptr));

    // Run simulation
    const float dt = T / NUM_STEPS;
    auto start = std::chrono::high_resolution_clock::now();

    gbm_kernel<<<blocks, BLOCK_SIZE>>>(d_paths, dt, d_states);
    CUDA_CHECK(hipDeviceSynchronize());

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    // Print results
    std::cout << "Simulated " << NUM_PATHS << " paths with " << NUM_STEPS << " steps each\n";
    std::cout << "Elapsed time: " << elapsed.count() << " seconds\n";
    std::cout << "Paths per second: " << NUM_PATHS / elapsed.count() << "\n";

    // Cleanup
    CUDA_CHECK(hipFree(d_paths));
    CUDA_CHECK(hipFree(d_states));
}

__global__ void curandSetupKernel(hiprandState* states, unsigned long seed) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= NUM_PATHS) return;
    hiprand_init(seed + idx, 0, 0, &states[idx]);
}

int main() {
    std::cout << "CUDA GBM Benchmark\n";
    std::cout << "GPU: " << []() {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
        return prop.name;
    }() << "\n";

    run_benchmark();
    return 0;
}